#include "hip/hip_runtime.h"
// Conjugate gradient function
#include "single_gpu_header.hpp"

// Cuda kernels
#include "single_gpu_kernels.cuh"

void conjugate_gradient(const double * A, double * x, const double * b, size_t size, int max_iters, double rel_error)
{
    // Declare the vectors used on the CPU

    double dot_temp;

    const unsigned int dot_product_threads = 256;
    const unsigned int dot_product_blocks = (size-1)/dot_product_threads + 1;
    const unsigned int gemv_tile_width = 16;
    const unsigned int gemv_blocks = (size - 1)/ gemv_tile_width + 1;
    const unsigned int gemv_threads = gemv_tile_width;
    const unsigned int axpby_threads = 256;
    const unsigned int axpby_blocks = (size-1)/axpby_threads + 1;
    const unsigned int matrix_bytes = sizeof(double) * size * size;
    const unsigned int vector_bytes = sizeof(double) * size;

    double* partial_dot = new double[dot_product_blocks];

    // First instantiate the vectors in the GPU memory
    double* dev_A;
    double* dev_b;
    double* dev_x;
    double* dev_Ap;
    double* dev_r;
    double* dev_p;
    double* dev_partial_dot;


    hipMalloc(&dev_A, matrix_bytes);
    hipMalloc(&dev_b, vector_bytes);
    hipMalloc(&dev_x, vector_bytes);
    hipMalloc(&dev_Ap, vector_bytes);
    hipMalloc(&dev_r, vector_bytes);
    hipMalloc(&dev_p, vector_bytes);
    hipMalloc(&dev_partial_dot, sizeof(double) * dot_product_blocks);


    // And copy data into the global memory of the GPU
    hipMemcpy(dev_A, A, matrix_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, vector_bytes, hipMemcpyHostToDevice);
    hipMemset(dev_x, 0, vector_bytes);
    hipMemcpy(dev_r, b, vector_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_p, b, vector_bytes, hipMemcpyHostToDevice);


    int num_iters;
    double alpha, beta, bb, rr, rr_new;

    // compute bb
    num_iters = 1;
    dot<<<dot_product_blocks, dot_product_threads, dot_product_threads * sizeof(double)>>>(
        dev_partial_dot,
        dev_b,
        dev_b,
        size
    );
    hipDeviceSynchronize();

    hipMemcpy(partial_dot, dev_partial_dot, sizeof(double) * dot_product_blocks, hipMemcpyDeviceToHost);
    bb = 0.0;
    
    #pragma unroll
    for (unsigned int i = 0; i < dot_product_blocks; ++i)
    {
        bb = bb + partial_dot[i];
    }
    rr = bb;

    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        // Compute A*p
        gemv<<<gemv_blocks, gemv_threads>>>(
            dev_A, 
            dev_p,
            dev_Ap, 
            size,
            size
        );
        hipDeviceSynchronize();


        // Compute the dot product (p, Ap)
        dot<<<dot_product_blocks, dot_product_threads, dot_product_threads * sizeof(double)>>>(
            dev_partial_dot,
            dev_p,
            dev_Ap,
            size
        );
        hipDeviceSynchronize();
        hipMemcpy(partial_dot, dev_partial_dot, sizeof(double) * dot_product_blocks, hipMemcpyDeviceToHost);
        dot_temp = 0.0;
        for (unsigned int i = 0; i < dot_product_blocks; ++i)
        {
            dot_temp = dot_temp + partial_dot[i];
        }
        alpha = rr / dot_temp;

        // Compute the correction of x with the residual

        axpby<<<axpby_blocks, axpby_threads>>>(
            alpha,
            dev_p,
            1.0,
            dev_x,
            size
        );
        hipDeviceSynchronize();

        // Compute the new residual vector

        axpby<<<axpby_blocks, axpby_threads>>>(
            -alpha, 
            dev_Ap, 
            1.0, 
            dev_r, 
            size
        );  
        hipDeviceSynchronize();

        // Compute the new norm of the residual
        dot<<<dot_product_blocks, dot_product_threads, dot_product_threads * sizeof(double)>>>(
            dev_partial_dot,
            dev_r,
            dev_r,
            size
        );
        hipDeviceSynchronize();
        hipMemcpy(partial_dot, dev_partial_dot, sizeof(double) * dot_product_blocks, hipMemcpyDeviceToHost);
        dot_temp = 0.0;
        for (unsigned int i = 0; i < dot_product_blocks; ++i)
        {
            dot_temp = dot_temp + partial_dot[i];
        }

        rr_new = dot_temp;
        beta = rr_new / rr;
        rr = rr_new;

        if(std::sqrt(rr / bb) < rel_error) { break; }
        axpby<<<axpby_blocks, axpby_threads>>>(
            1.0,
            dev_r,
            beta,
            dev_p, 
            size
        );
        hipDeviceSynchronize();
    }

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
    hipMemcpy(x, dev_x, sizeof(double) * size, hipMemcpyDeviceToHost);
    // All Cuda free that will eventually come

}


