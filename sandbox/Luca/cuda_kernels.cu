// Include (.cu) with the implementation of the kernel
#include "cuda_kernels.cuh"

// Include (.hpp) used to link with the .cpp file
#include "gpu_tests.hpp"

void daxpy_func(double a, double* x, double* y, unsigned int size)
{
    // just compute the stride and 
    // call the kernel.

    unsigned int num_blocks = 25;
    unsigned int thread_per_blocks = 512;

    // allocate device memory
    double* dev_x;
    hipMalloc(&dev_x, size * sizeof(double));
    double* dev_y;
    hipMalloc(&dev_y, size * sizeof(double));

    hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, size * sizeof(double), hipMemcpyHostToDevice);
    std::cout << "Calling the kernel" << std::endl;

    double c = 1.0;

    saxpy_kernel<<<num_blocks, thread_per_blocks>>>(c, dev_x, dev_y, size);
    hipDeviceSynchronize();
    std::cout << "Kernel returned" << std::endl;
    hipMemcpy(x, dev_x, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_x);
    hipFree(dev_y);
}

double vec_dot_func_optimized(double* a, double* b, unsigned int size)
{
    unsigned int num_blocks = 25;
    unsigned int threads_per_block = 1024;

    // allocate device memory
    double* dev_a;
    hipMalloc(&dev_a, size * sizeof(double));
    double* dev_b;
    hipMalloc(&dev_b, size * sizeof(double));
   
    double* res;
    res = (double*) malloc(num_blocks * sizeof(double));
    double* dev_res;
    hipMalloc(&dev_res, num_blocks * sizeof(double));

    std::cout << std::endl;
    // copy data to the device
    hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);

    vec_dot_kernel_optimized<<<num_blocks, threads_per_block, threads_per_block * sizeof(double)>>>(dev_a, dev_b, size, dev_res);
    hipDeviceSynchronize();

    hipMemcpy(res, dev_res, num_blocks * sizeof(double), hipMemcpyDeviceToHost);
    double dot_product = 0;

    for(unsigned int i = 0; i < num_blocks; i++)
    {
        dot_product = dot_product + res[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    free(res);

    return dot_product;
}

void A_times_x_func(double* A,
    double* x, 
    double* res, 
    unsigned int num_rows, 
    unsigned int num_cols)
{
    unsigned int num_blocks = 25;
    unsigned int num_threads = 1024;
    
    double* dev_A;
    double* dev_x;

    double* dev_res;

    // allocating all the vector on the gpu

    unsigned int total_elems = num_rows * num_cols;

    std::cout << "Allocating gpu memory...";

    hipMalloc(&dev_A, total_elems * sizeof(double));
    hipMalloc(&dev_x, (num_rows) * sizeof(double));
    hipMalloc(&dev_res, (num_rows) * sizeof(double));

    std::cout << "done" << std::endl;
    // copying data from cpu to gpu

    std::cout << "Copying data towards gpu...";
    hipMemcpy(dev_A, A, total_elems * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, num_rows * sizeof(double), hipMemcpyHostToDevice);
    std::cout << "done" << std::endl;


    unsigned int num_blocks = 3;
    unsigned int threads_per_block = 1024;

    std::cout << "Calling the kernel...";
    A_times_x_kernel<<<num_blocks, threads_per_block>>>(
        dev_A,
        dev_x,
        dev_res,
        num_rows,
        num_cols,
    )
    hipDeviceSynchronize();
    std::cout << "done";

    hipMemcpy(res, dev_res, num_rows * sizeof(double), hipMemcpyDeviceToHost);
    
}

