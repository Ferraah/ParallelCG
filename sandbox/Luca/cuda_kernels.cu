// Include (.cu) with the implementation of the kernel
#include "cuda_kernels.cuh"

// Include (.hpp) used to link with the .cpp file
#include "gpu_tests.hpp"

void vec_sum_func(double* a, double* b, unsigned int size)
{
    // just compute the stride and 
    // call the kernel.

    unsigned int num_blocks = 25;
    unsigned int thread_per_blocks = 512;

    // allocate device memory
    double* dev_a;
    hipMalloc(&dev_a, size * sizeof(double));
    double* dev_b;
    hipMalloc(&dev_b, size * sizeof(double));

    hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    std::cout << "Calling the kernel" << std::endl;
    vec_sum_kernel<<<num_blocks, thread_per_blocks>>>(dev_a, dev_b, size);
    hipDeviceSynchronize();
    std::cout << "Kernel returned" << std::endl;
    hipMemcpy(a, dev_a, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
}

double vec_dot_func(double* a, double* b, unsigned int size)
{
    unsigned int num_blocks = 25;
    unsigned int threads_per_block = 1024;

    // allocate device memory
    std::cout << "Allocating memory on the device" << std::endl;
    double* dev_a;
    hipMalloc(&dev_a, size * sizeof(double));
    double* dev_b;
    hipMalloc(&dev_b, size * sizeof(double));
   
    double* res;
    res = (double*) malloc(num_blocks * sizeof(double));
    double* dev_res;
    hipMalloc(&dev_res, num_blocks * sizeof(double));

    std::cout << std::endl;
    // copy data to the device
    hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);

    std::cout << "Calling the kernel" << std::endl;
    vec_dot_kernel<<<num_blocks, threads_per_block, threads_per_block * sizeof(double)>>>(dev_a, dev_b, size, dev_res, threads_per_block);
    hipDeviceSynchronize();

    hipMemcpy(res, dev_res, num_blocks * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Computing the reduction" << std::endl;
    
    double dot_product = 0;

    for(unsigned int i = 0; i < num_blocks; i++)
    {
        dot_product = dot_product + res[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    free(res);

    return dot_product;
}